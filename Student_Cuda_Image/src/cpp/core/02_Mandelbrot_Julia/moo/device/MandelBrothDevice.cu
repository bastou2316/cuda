#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "math/MandelBrothMath.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelBroth(uchar* ptrDevPixels, uint w, uint h, uint stopIteration, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void mandelBroth(uchar4* ptrDevPixels, uint w, uint h, uint stopIteration, DomaineMath domaineMath)
	    {
		MandelBrothMath mandelBrothMath = MandelBrothMath(stopIteration);

		const int TID = Indice2D::tid();
		const int NB_THREAD = Indice2D::nbThread();
		const int WH = w * h ;

		int pixelI;
		int pixelJ;

		int s = TID;
		double x,y;
		while( s < WH)
		    {
		    IndiceTools::toIJ(s, w, &pixelI, &pixelJ);


		    domaineMath.toXY(pixelI, pixelJ, &x, &y);
		    mandelBrothMath.colorXY(&ptrDevPixels[s], (float)x, (float)y);
		    s+= NB_THREAD;

		    }
	    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
