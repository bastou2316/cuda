#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "math/JuliaMath.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void julia(uchar* ptrDevPixels, uint w, uint h, uint stopIteration, float realJuliaParameter, float imaginaryJuliaParameter, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void julia(uchar4* ptrDevPixels, uint w, uint h, uint stopIteration, float realJuliaParameter, float imaginaryJuliaParameter, DomaineMath domaineMath)
	    {
		JuliaMath juliaMath = JuliaMath(stopIteration, realJuliaParameter, imaginaryJuliaParameter);

		const int TID = Indice2D::tid();
		const int NB_THREAD = Indice2D::nbThread();
		const int WH = w * h ;
		int pixelI;
		int pixelJ;

		int s = TID;
		double x,y;
		while( s < WH)
		    {
		    IndiceTools::toIJ(s, w, &pixelI, &pixelJ);


		    domaineMath.toXY(pixelI, pixelJ, &x, &y);
		    juliaMath.colorXY(&ptrDevPixels[s], (float)x, (float)y);
		    s+= NB_THREAD;

		    }
	    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
