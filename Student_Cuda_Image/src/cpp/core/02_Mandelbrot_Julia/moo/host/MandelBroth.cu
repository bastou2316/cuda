#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "Device.h"
#include "MandelBroth.h"
#include "Fractale.h"

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
/*
 * IMPORTED
 *
 * */

extern __global__ void mandelBroth(uchar4* ptrDevPixels, uint w, uint h, uint n, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

MandelBroth::MandelBroth(const Grid& grid, uint w, uint h, uint n, const DomaineMath& domaineMath) : Fractale (grid, w, h, n, domaineMath)
    {
	this->n = n;
    }
MandelBroth::~MandelBroth()
    {

    }

void MandelBroth::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
	Device::lastCudaError("Mandelbroth BEFORE");
	mandelBroth<<<dg,db>>>(ptrDevPixels, w, h, n, domaineMath);
	Device::lastCudaError("mandelbroth AFTER");
    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
