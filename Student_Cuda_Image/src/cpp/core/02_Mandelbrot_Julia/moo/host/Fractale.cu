#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "Device.h"
#include "Fractale.h"

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
/*
 * IMPORTED
 *
 * */

extern __global__ void fractale(uchar4* ptrDevPixels, uint w, uint h, uint n, float t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Fractale::Fractale(const Grid& grid, uint w, uint h, uint n, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Fractale CUDA", domaineMath)
    {
    this->n = n;
    this->t = 0.0f;
    }
Fractale::~Fractale()
    {

    }

void Fractale::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
	//TODO : CLASSE FILLES
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
