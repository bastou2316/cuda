#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "Device.h"
#include "Julia.h"
#include "Fractale.h"

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
/*
 * IMPORTED
 *
 * */

extern __global__ void julia(uchar4* ptrDevPixels, uint w, uint h, uint n, float realJuliaParameter, float imaginaryJuliaParameter, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Julia::Julia(const Grid& grid, uint w, uint h, uint n, float realJuliaParameter, float imaginaryJuliaParameter, const DomaineMath& domaineMath) :
	Fractale(grid, w, h, n, domaineMath)
    {
    this->realJuliaParameter = realJuliaParameter;
    this->imaginaryJuliaParameter = imaginaryJuliaParameter;
    }
Julia::~Julia()
    {

    }

void Julia::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
	Device::lastCudaError("Mandelbroth BEFORE");
    	julia<<<dg,db>>>(ptrDevPixels, w, h, n, realJuliaParameter, imaginaryJuliaParameter, domaineMath);
    	Device::lastCudaError("mandelbroth AFTER");
    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
