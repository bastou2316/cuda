
#include <hip/hip_runtime.h>
//#include "Indice2D.h"
//#include "cudaTools.h"
//#include "Device.h"
//
//#include "IndiceTools_GPU.h"
//
//#include "RayTracingMath.h"
//using namespace gpu;
//
//// Attention : 	Choix du nom est impotant!
////		VagueDevice.cu et non Vague.cu
//// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
////		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!
//
///*----------------------------------------------------------------------*\
// |*			Declaration 					*|
// \*---------------------------------------------------------------------*/
//
///*--------------------------------------*\
// |*		Imported	 	*|
// \*-------------------------------------*/
//
///*--------------------------------------*\
// |*		Public			*|
// \*-------------------------------------*/
//
//__global__ void RayTracing(uchar4* ptrDevPixels, uint w, uint h, float t);
//
///*--------------------------------------*\
// |*		Private			*|
// \*-------------------------------------*/
//
///*----------------------------------------------------------------------*\
// |*			Implementation 					*|
// \*---------------------------------------------------------------------*/
//
///*--------------------------------------*\
// |*		Public			*|
// \*-------------------------------------*/
//
//__global__ void RayTracing(uchar4* ptrDevPixels, uint w, uint h, float t)
//    {
//    RayTracingMath RayTracingMath = RayTracingMath(w, h);
//
//    const int TID = Indice2D::tid();
//    const int NB_THREAD = Indice2D::nbThread();
//    const int WH = w * h;
//
//    // TODO pattern entrelacement
//    int s = TID;
//
//    int pxI;
//    int pxJ;
//
//    while(s < WH)
//	{
//	IndiceTools::toIJ(s,w,&pxI,&pxJ);
//
//	RayTracingMath.colorXY(&ptrDevPixels[s],pxI, pxJ, t);
//	//ptrDevPixels[s] = color;
//	s += NB_THREAD;
//	}
//    }
//
///*--------------------------------------*\
// |*		Private			*|
// \*-------------------------------------*/
//
///*----------------------------------------------------------------------*\
// |*			End	 					*|
// \*---------------------------------------------------------------------*/
//
