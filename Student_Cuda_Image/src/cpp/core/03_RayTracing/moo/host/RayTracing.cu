
#include <hip/hip_runtime.h>
//#include <iostream>
//#include <assert.h>
//
//#include "Device.h"
//#include "Rippling.h"
//#include <assert.h>
//
//using std::cout;
//using std::endl;
//
///*----------------------------------------------------------------------*\
// |*			Declaration 					*|
// \*---------------------------------------------------------------------*/
//
///*--------------------------------------*\
// |*		Imported	 	*|
// \*-------------------------------------*/
//
//extern __global__ void RayTracing(uchar4* ptrDevPixels,uint w, uint h,float t);
//
///*--------------------------------------*\
// |*		Public			*|
// \*-------------------------------------*/
//
///*--------------------------------------*\
// |*		Private			*|
// \*-------------------------------------*/
//
///*----------------------------------------------------------------------*\
// |*			Implementation 					*|
// \*---------------------------------------------------------------------*/
//
///*--------------------------------------*\
// |*		Public			*|
// \*-------------------------------------*/
//
///*-------------------------*\
// |*	Constructeur	    *|
// \*-------------------------*/
//
//RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt) :
//	Animable_I<uchar4>(grid, w, h, "RayTracing_Cuda_RGBA_uchar4")
//    {
//    assert(w == h); // specific rippling
//
//    // Inputs
//    this->dt = dt;
//
//    // Tools
//    this->t = 0; // protected dans Animable
//
//    this.spheres = new Spheres[n];
//    //Boucle ici
//    }
//
//RayTracing::~RayTracing()
//    {
//    // rien
//    }
//
///*-------------------------*\
// |*	Methode		    *|
// \*-------------------------*/
//
///**
// * Override
// * Call periodicly by the API
// *
// * Note : domaineMath pas use car pas zoomable
// */
//void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
//    {
//    Device::lastCudaError("rippling rgba uchar4 (before)"); // facultatif, for debug only, remove for release
//
//    // TODO lancer le kernel avec <<<dg,db>>>
//    rayTracing <<<dg,db>>> (ptrDevPixels,w,h,t);
//    // le kernel est importer ci-dessus (ligne 19)
//
//    Device::lastCudaError("rippling rgba uchar4 (after)"); // facultatif, for debug only, remove for release
//    }
//
///**
// * Override
// * Call periodicly by the API
// */
//void RayTracing::animationStep()
//    {
//    t += dt;
//    }
//
///*--------------------------------------*\
// |*		Private			*|
// \*-------------------------------------*/
//
///*----------------------------------------------------------------------*\
// |*			End	 					*|
// \*---------------------------------------------------------------------*/
//
